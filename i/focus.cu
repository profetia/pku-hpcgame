
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>

#define CEIL(a, b) (((a) + (b) - 1) / (b))

#define FLOAT4(a) *((float4*)&(a))
#define DOUBLE4(a) *((double4*)&(a))

using Point = double3;

static __device__ __forceinline__ double distance(Point a, Point b) {
  double dx = a.x - b.x;
  double dy = a.y - b.y;
  double dz = a.z - b.z;
  return sqrt(dx * dx + dy * dy + dz * dz);
}

constexpr int WARP_SIZE = 32;
constexpr int BLOCK_SIZE = WARP_SIZE;
constexpr int WINDOW_SIZE = 8;

__global__ void focus_ab(const Point kSrc, const Point* mirs, const int kMirN,
                         const Point* sens, const int kSenN, float* a,
                         float* b) {
  int kMirOffset = blockIdx.x * blockDim.x * WINDOW_SIZE;
  int kSenOffset = blockIdx.y * blockDim.y * WINDOW_SIZE;
  int kWarpIdx = threadIdx.y;
  int kWarpLane = threadIdx.x;

  __shared__ Point local_mirs[BLOCK_SIZE * WINDOW_SIZE];
  __shared__ Point local_sens[BLOCK_SIZE * WINDOW_SIZE];

  if (kWarpIdx < WINDOW_SIZE) {
    int kMirLocalIdx = kWarpIdx * BLOCK_SIZE + kWarpLane;
    local_mirs[kMirLocalIdx] = mirs[kMirOffset + kMirLocalIdx];
  } else if (kWarpIdx < WINDOW_SIZE * 2) {
    int kSenLocalIdx = (kWarpIdx - WINDOW_SIZE) * BLOCK_SIZE + kWarpLane;
    local_sens[kSenLocalIdx] = sens[kSenOffset + kSenLocalIdx];
  }

  __syncthreads();

  float local_a[WINDOW_SIZE] = {0};
  float local_b[WINDOW_SIZE] = {0};

  for (int i = 0; i < WINDOW_SIZE; ++i) {
    Point sen = local_sens[kWarpIdx * WINDOW_SIZE + i];
#pragma unroll
    for (int j = 0; j < WINDOW_SIZE; ++j) {
      Point mir = local_mirs[kWarpLane * WINDOW_SIZE + j];
      double l = distance(mir, kSrc) + distance(mir, sen);
      double a, b;
      sincos(6.283185307179586 * 2000 * l, &b, &a);

      local_a[i] += static_cast<float>(a);
      local_b[i] += static_cast<float>(b);
    }

#pragma unroll
    for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2) {
      local_a[i] += __shfl_down_sync(0xffffffff, local_a[i], offset);
      local_b[i] += __shfl_down_sync(0xffffffff, local_b[i], offset);
    }
  }

  if (kWarpLane == 0) {
#pragma unroll
    for (int i = 0; i < WINDOW_SIZE; ++i) {
      atomicAdd(&a[kSenOffset + kWarpIdx * WINDOW_SIZE + i], local_a[i]);
      atomicAdd(&b[kSenOffset + kWarpIdx * WINDOW_SIZE + i], local_b[i]);
    }
  }
}

__global__ void focus_illum(const float* a, const float* b, const int kSenN,
                            double* illums) {
  int kSenIdx = (blockIdx.x * blockDim.x + threadIdx.x) * 4;
  if (kSenIdx < kSenN) {
    float4 a4 = FLOAT4(a[kSenIdx]);
    float4 b4 = FLOAT4(b[kSenIdx]);

    double4 illum4;
    illum4.x = static_cast<double>(sqrt(a4.x * a4.x + b4.x * b4.x));
    illum4.y = static_cast<double>(sqrt(a4.y * a4.y + b4.y * b4.y));
    illum4.z = static_cast<double>(sqrt(a4.z * a4.z + b4.z * b4.z));
    illum4.w = static_cast<double>(sqrt(a4.w * a4.w + b4.w * b4.w));

    DOUBLE4(illums[kSenIdx]) = illum4;
  }
}

int main() {
  Point kSrc;
  int64_t kMirN, kSenN;
  Point *mirs, *sens;

  {
    FILE* infile = fopen("in.data", "rb");
    fread(&kSrc, sizeof(Point), 1, infile);

    fread(&kMirN, sizeof(int64_t), 1, infile);
    hipHostAlloc(&mirs, kMirN * sizeof(Point), hipHostMallocDefault);
    fread(mirs, sizeof(Point), kMirN, infile);

    fread(&kSenN, sizeof(int64_t), 1, infile);
    hipHostAlloc(&sens, kSenN * sizeof(Point), hipHostMallocDefault);
    fread(sens, sizeof(Point), kSenN, infile);

    fclose(infile);
  }

  double* illums;
  hipHostAlloc(&illums, kSenN * sizeof(double), hipHostMallocDefault);

  Point *d_mirs, *d_sens;
  hipMalloc(&d_mirs, kMirN * sizeof(Point));
  hipMalloc(&d_sens, kSenN * sizeof(Point));

  float *d_a, *d_b;
  hipMalloc(&d_a, kSenN * sizeof(float));
  hipMalloc(&d_b, kSenN * sizeof(float));

  double* d_illums;
  hipMalloc(&d_illums, kSenN * sizeof(double));

  hipMemcpy(d_mirs, mirs, kMirN * sizeof(Point), hipMemcpyHostToDevice);
  hipMemcpy(d_sens, sens, kSenN * sizeof(Point), hipMemcpyHostToDevice);
  hipMemset(d_a, 0, kSenN * sizeof(float));
  hipMemset(d_b, 0, kSenN * sizeof(float));

  {
    dim3 block_dim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid_dim(CEIL(CEIL(kMirN, WINDOW_SIZE), BLOCK_SIZE),
                  CEIL(CEIL(kSenN, WINDOW_SIZE), BLOCK_SIZE));

    focus_ab<<<grid_dim, block_dim>>>(kSrc, d_mirs, kMirN, d_sens, kSenN, d_a,
                                      d_b);
  };

  {
    int block_dim = 1024;
    int grid_dim = CEIL(CEIL(kSenN, 4), block_dim);

    focus_illum<<<grid_dim, block_dim>>>(d_a, d_b, kSenN, d_illums);
  }

  hipMemcpy(illums, d_illums, kSenN * sizeof(double), hipMemcpyDeviceToHost);

  {
    FILE* outfile = fopen("out.data", "wb");
    fwrite(illums, sizeof(double), kSenN, outfile);
    fclose(outfile);
  }

  hipFree(d_mirs);
  hipFree(d_sens);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_illums);

  hipHostFree(mirs);
  hipHostFree(sens);
  hipHostFree(illums);

  return 0;
}